#include "hip/hip_runtime.h"
#include "gpgpu_cuda.cuh"

#include "morphology/StrEl.h"

#include <opencv2/core/mat.hpp>

__global__ void kdilate(uint8_t *pIn, uint8_t *pOut, uint8_t *pSe, uint8_t *pM, int imW, int imH, int seW, cv::Point seC) {
    uint32_t x = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint32_t y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (y >= imH || x >= imW) return;

    int val = 0;

    for (int j = -seC.y; j <= seC.y; ++j) {
        for (int i = -seC.x; i <= seC.x; ++i) {
            int seIdx = (seC.y + j) * seW + seC.x + i;
            if (pM[seIdx] == 0) continue;
            if (y + j >= imH || x + i >= imW) continue;

            int imIdx = (y + j) * imW + x + i;
            val = max(val, pIn[imIdx] + pSe[seIdx]);
        }
    }

    pOut[y * imW + x] = static_cast<uint8_t>(min(val, 0xFF));
}

cv::Mat morphcuda::dilate(const GridConfig &gridConfig, cv::Mat &img, StrEl &strEl) {
    assert(img.type() == CV_8UC1);

    hipEvent_t evtStart, evtCopy, evtCompute, evtCopyBack;
    hipEventCreate(&evtStart);
    hipEventCreate(&evtCopy);
    hipEventCreate(&evtCompute);
    hipEventCreate(&evtCopyBack);

    uint8_t *d_img, *d_out, *d_strel, *d_mask;

    int imgSz = img.rows * img.cols * sizeof(uint8_t);
    int strelSz = strEl.size().width * strEl.size().height * sizeof(uint8_t);    // assuming StrEl has expansion of 1

    cv::Mat out(img.size(), CV_8UC1);

    hipEventRecord(evtStart);

    // Consider using hipMallocManaged
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&d_img), imgSz));
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&d_out), imgSz));
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&d_strel), strelSz));
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&d_mask), strelSz));

    gpuErrchk(hipMemcpy(d_img, img.data, imgSz, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_strel, strEl.elemPtr(), strelSz, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_mask, strEl.maskPtr(), strelSz, hipMemcpyHostToDevice));

    hipEventRecord(evtCopy);
    kdilate<<<gridConfig.gridBlocksSz, gridConfig.blockThreadsSz>>>(
            d_img, d_out, d_strel, d_mask, img.size().width, img.size().height, strEl.size().width, cv::Point(strEl.xMax(),strEl.yMax()));
    hipEventRecord(evtCompute);

    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(out.data, d_out, imgSz, hipMemcpyDeviceToHost));

    hipEventRecord(evtCopyBack);

    gpuErrchk(hipFree(d_img));
    gpuErrchk(hipFree(d_out));
    gpuErrchk(hipFree(d_strel));
    gpuErrchk(hipFree(d_mask));

    float timeCopy, timeExec, timeCopyBack;
    hipEventElapsedTime(&timeCopy, evtStart, evtCopy);
    hipEventElapsedTime(&timeExec, evtCopy, evtCompute);
    hipEventElapsedTime(&timeCopyBack, evtCompute, evtCopyBack);

    fprintf(stderr, "Done in %fms (%fms copy, %fms exec, %fms copy back)\n",
            timeCopy + timeExec + timeCopyBack, timeCopy, timeExec, timeCopyBack);

    return out;
}
